
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define N 9999     // number of bodies
#define MASS 0     // row in array for mass
#define X_POS 1    // row in array for x position
#define Y_POS 2    // row in array for y position
#define Z_POS 3    // row in array for z position
#define X_VEL 4    // row in array for x velocity
#define Y_VEL 5    // row in array for y velocity
#define Z_VEL 6    // row in array for z velocity
#define G 200      // "gravitational constant" (not really)
#define MU 0.001   // "frictional coefficient"
#define BOXL 100.0 // periodic boundary box length
#define dt 0.05  // time interval
#define W 7

float body[10000][7]; // data array of bodies


__global__ void init(unsigned int seed, hiprandState_t* states, int n) {

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
  hiprand_init(seed,
              i,
              0,
              &states[i]);
    }
}
__device__ void crossProduct(float vect_A[], float vect_B[], float cross_P[]) { 
  cross_P[0] = vect_A[1] * vect_B[2] - vect_A[2] * vect_B[1]; 
  cross_P[1] = vect_A[2] * vect_B[0] - vect_A[0] * vect_B[2]; 
  cross_P[2] = vect_A[0] * vect_B[1] - vect_A[1] * vect_B[0]; 
}
__device__ void norm(float &x, float &y, float &z) {
  float mag = sqrt(x*x+y*y+z*z);
  x/=mag; y/=mag; z/=mag;
}
__global__ void randoms (hiprandState_t* states, float* boddy, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    float vect_A[3], vect_B[3], cross_P[3];

    if (i < n && i>0) {
      // get x, y, and z positions
      boddy[i* W+MASS] = 0.001; // 0
      boddy[i* W+X_POS] = (hiprand_uniform(&states[i]))*300 - (150+0.999); // 1
      boddy[i* W+Y_POS] = (hiprand_uniform(&states[i]))* 300 -(150+0.999); // 2
      boddy[i* W+Z_POS] = (hiprand_uniform(&states[i]))*300 -(150+0.999); // 3
      // compute norm
      vect_A[0]= boddy[i* W+X_POS];
      vect_A[1]= boddy[i* W+Y_POS];
      vect_A[2]= boddy[i* W+Z_POS];
      norm(vect_A[0], vect_A[1], vect_A[2]);
      // get the cross product
      vect_B[0]= 0.0; vect_B[1]= 0.0; vect_B[2]= 1.0;
      cross_P[0] = 0.0; cross_P[1] = 0.0; cross_P[2] = 0.0; 
      crossProduct(vect_A, vect_B, cross_P);

       // random initial velocities magnitudes
      boddy[i*W+X_VEL] = (hiprand_uniform(&states[i]))*(100+0.999) *cross_P[0];
      boddy[i*W+Y_VEL] = (hiprand_uniform(&states[i]))*(100+0.999) *cross_P[1];
      boddy[i*W+Z_VEL] = (hiprand_uniform(&states[i]))*(100+0.999) *cross_P[2];


    }

}
__global__ void forces(float* dev_body, float* dev_fx, float* dev_fy, float* dev_fz, int n){

  float x_diff, y_diff, z_diff; 
  int i = threadIdx.x + blockIdx.x * blockDim.x; 

  for(int x = 0 ; x < n; x++){
    if(x != i && i < n){
      x_diff = dev_body[i*W+X_POS] - dev_body[x*W+X_POS];
      y_diff = dev_body[i*W+Y_POS] - dev_body[x*W+Y_POS];
      z_diff = dev_body[i*W+Z_POS] - dev_body[x*W+Z_POS];
  
	    // calculate distance (r)
      float rr = (x_diff * x_diff + y_diff * y_diff + z_diff * z_diff);
      float r = sqrt(rr);

      // force between bodies i and x
      float F = 0;

      // if sufficiently far away, apply gravitation force
      if (r > 50.0) {
        // TODO: compute gravitational force between body i and x
        F = -1.0*(G*dev_body[i*W+MASS]*dev_body[x*W+MASS]) / rr; 
        norm(x_diff, y_diff, z_diff); 
        dev_fx[i] += (x_diff/r)*F; 
        dev_fy[i] += (y_diff/r)*F; 
        dev_fz[i] += (z_diff/r)*F; 
      } 
    }
  }
}

__global__ void update(float* dev_body, float* dev_fx, float* dev_fy, float* dev_fz, int n){

  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < n) {

    dev_body[i*W+X_VEL] = dev_body[i*W+X_VEL] + dev_fx[i]*dt / dev_body[i*W+MASS]; 
    dev_body[i*W+Y_VEL] = dev_body[i*W+Y_VEL] + dev_fy[i]*dt / dev_body[i*W+MASS]; 
    dev_body[i*W+Z_VEL] = dev_body[i*W+Z_VEL] + dev_fz[i]*dt / dev_body[i*W+MASS]; 

    // TODO: update positions
    dev_body[i*W+X_POS] = dev_body[i*W+X_POS] + dev_body[i*W+X_VEL]*dt; 
    dev_body[i*W+Y_POS] = dev_body[i*W+Y_POS] + dev_body[i*W+Y_VEL]*dt; 
    dev_body[i*W+Z_POS] = dev_body[i*W+Z_POS] + dev_body[i*W+Z_VEL]*dt;
  } 

}

__global__ void initForce(float* dev_fx,float*  dev_fy,float*  dev_fz, int n){
  int i = threadIdx.x + blockDim.x*blockIdx.x; 
  if(i < n){
    dev_fx[i] = 0; 
    dev_fy[i] = 0; 
    dev_fz[i] = 0; 
  }
}


int main(int argc, char **argv) {

  int tmax = 0;
  float Fx_dir[N], Fy_dir[N], Fz_dir[N];
  float * dev_fx, *dev_fy, *dev_fz;

  if (argc != 2) {
    fprintf(stderr, "Format: %s { number of timesteps }\n", argv[0]);
    exit (-1);
  }

  tmax = atoi(argv[1]);
  dim3 dimBlock(1024);
  dim3 dimGrid((int)ceil((float)N / 1024)); 

  if(tmax<0){
    fprintf(stderr, "No negative values for time allowed\n");

  }

  // assign each body a random initial positions and velocities
  // black hole at the center
  body[0][MASS] = 4000.0; body[0][X_POS] = 0.0;body[0][Y_POS] = 0.0;
  body[0][Z_POS] = 0.0;body[0][X_VEL] = 0.0;body[0][Y_VEL] = 0.0;body[0][Z_VEL] = 0.0;
  
// PARALLELIZED THE RANDOM INITIALIZATION OF MASS AND POSITIONS

  hiprandState_t* states;
  // allocate space on GPU for random states
  hipMalloc((void**) &states, N*7*sizeof(hiprandState_t));
  
  /* invoke the GPU to initialize all of the random states */
  init<<<dimGrid, dimBlock>>>(time(0), states, N*7);
  hipDeviceSynchronize();

  // allocate array of unsigned ints on CPU and GPU
  float* body_arr;
  hipMalloc((void**) &body_arr, N*7*sizeof(float));

  // obtain a uniformly random distriubtion of integers, maximum N
  randoms<<<dimGrid, dimBlock>>>(states, body_arr, N*7);
  hipDeviceSynchronize();
  hipMemcpy(body, body_arr, N*7*sizeof(float), hipMemcpyDeviceToHost);
// finished coppying all the initial values to body

  // print out initial positions in PDB format
  printf("MODEL %8d\n", 0);
  for (int i = 0; i < N; i++) {
    printf("%s%7d  %s %s %s%4d    %8.3f%8.3f%8.3f  %4.2f  %4.3f\n",
           "ATOM", i+1, "CA ", "GLY", "A", i+1, body[i][X_POS], body[i][Y_POS], body[i][Z_POS], 1.00, 0.00);
  }
  printf("TER\nENDMDL\n");


// FORCE CALCULATION !!!!

  hipMalloc((void**) &dev_fx, N*sizeof(float));
  hipMalloc((void**) &dev_fy, N*sizeof(float));
  hipMalloc((void**) &dev_fz, N*sizeof(float));

  hipEvent_t start;
  hipEventCreate(&start);
  hipEvent_t stop;
  hipEventCreate(&stop);

  // start timer
  hipEventRecord(start,0);

  for (int t = 0; t < tmax; t++) {
    // TODO: initialize forces to zero

    for(int i = 0; i < N; i++) {
      Fx_dir[i] = 0.0; 
      Fy_dir[i] = 0.0; 
      Fz_dir[i] = 0.0; 
      
    }

    // initForce<<<dimGrid, dimBlock>>>(dev_fx,dev_fy,dev_fz, N); 
    hipMemcpy(dev_fx, Fx_dir, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_fy, Fy_dir, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_fz, Fz_dir, sizeof(float), hipMemcpyHostToDevice);

    // cudaDeviceSynchronize(); 

    forces<<<dimGrid, dimBlock>>>(body_arr, dev_fx,dev_fy,dev_fz, N); 
    hipDeviceSynchronize();

    update<<<dimGrid, dimBlock>>>(body_arr, dev_fx, dev_fy, dev_fz, N);
    hipDeviceSynchronize();
    hipMemcpy(body, body_arr, N*7*sizeof(float), hipMemcpyDeviceToHost);

    // print out positions in PDB format
    printf("MODEL %8d\n", t+1);
    for (int i = 0; i < N; i++) {
	printf("%s%7d  %s %s %s%4d    %8.3f%8.3f%8.3f  %4.2f  %4.3f\n",
               "ATOM", i+1, "CA ", "GLY", "A", i+1, body[i][X_POS], body[i][Y_POS], body[i][Z_POS], 1.00, 0.00);
    }
    printf("TER\nENDMDL\n");

  }
  
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);

  float diff;
  hipEventElapsedTime(&diff, start, stop);
  printf("time: %f ms\n", diff);

  // deallocate timers
  hipEventDestroy(start);
  hipEventDestroy(stop);


  hipFree(states);
  hipFree(body_arr);
  hipFree(dev_fx); 
  hipFree(dev_fy); 
  hipFree(dev_fz); 
}
